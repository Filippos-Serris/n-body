#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define G 6.673e-11 // gravity constant
#define M 100       // body mass
#define Dt 0.03     // time between iterations
#define Exp 0.01    // softening parameter

__global__ void newPos(int *loop,int *N,double *C,double *V,double *F)
{
  int i,index,k;

  // dx/dy/dz: αποστάσεις ανα άξονα (x,y,z)
  // d: απόσταση δύο σωμάτων
  double dx,dy,dz,d;

  // dSquare: τετράγωνο απόστασης μεταξύ σωμάτων
  // mSquare: τετράγωνο μάζας σωμάτων
  // expSquare: τετράγωνο μεαβλητής απόσβεσης
  double dSquare,mSquare,expSquare;

  // a: επιτάχυνση σώματος
  double a;

  index=threadIdx.x+blockIdx.x*blockDim.x;

  mSquare=pow(M,2);
  expSquare=pow(Exp,2);

  // Υπολογισμοί
  for(i=0;i<*loop;i++)            // i: επαναλήψης πειράματως
  {
    if(index<*N)                  // index: σώμα στο οποίο ασκουντε δυνάμεις
    {
      for(k=0;k<*N;k++)           // k: σώμα το οποίο ασκεί δύναμη στο σώμα j
      {
        if(index!=k)
        {
          dx=C[k*3+0]-C[index*3+0];
          dy=C[k*3+1]-C[index*3+1];
          dz=C[k*3+2]-C[index*3+2];

          d=sqrt(pow(dx,2)+pow(dy,2)+pow(dz,2));
          dSquare=pow(d,2);

          F[index*3+0]-=G*mSquare*d/(pow(dSquare+expSquare,1.5)*dx);            // Αθροιστική δύναμη που ασκείται στο σώμα j
          F[index*3+1]-=G*mSquare*d/(pow(dSquare+expSquare,1.5)*dy);            // απο όλα τα υπόλοιπα σώματα k του συστήματος
          F[index*3+2]-=G*mSquare*d/(pow(dSquare+expSquare,1.5)*dz);            // στους τρίς άξονες (x,y,z)
        }
      }
    }

    if(index<*N)                                                // index: σώμα για το οποίο υπολογίζονται οι νέες συντεταγμένες
    {
      for(k=0;k<3;k++)                                          // k: συντεταγμένη που υπολογίζεται σε καθε κύκλο (x,y,z)
      {
        a=F[index*3+k]/M;                                       // Επιτάχυνση α=F/M

        F[index*3+k]=0.0;                                       // Επαναφορά της δύναμης F σε 0

        V[index*3+k]=V[index*3+k]+a*Dt;                         // Ταχύτητα V(n+1)=Vn+α*Δt
        C[index*3+k]=C[index*3+k]+V[index*3+k]*Dt;              // Θέση R(n+1)=Rn+Vn*Δt
      }
    }
  }
}

int main()
{
  // Μεταβλητές host
  int i;

  // h_N: πλήθος σωμάτων, h_loop: πλήθος επαναλήψεων
  int h_N,h_loop;

  // u/v: μεταβλητές ορισμού κανονικής κατανομής του x
  // x1: τιμή μεταξύ του [0,1] που ακολουθεί την κανονική κατανομή
  // r: μεταβλητή προσδιορισμού θέσης
  double u,v,x1,r;

  // x4/x5: τυχαίες μεταβλητές για τον ορισμό της vel
  // vesc: αρχική ταχύτητα (ταχύτητα αποφυγής)
  // vel: μεταβλητή προσδιορισμού ταχύτητας
  double x4,x5,vel,vesc;

  // theta: τυχαία τιμή μεταξύ [0,π], phi: τυχαία τιμή μεταξύ [0,2π]
  double theta,phi;

  // C: πίνακας συντεταγμένων, V: πίνακας ταχυτήτων, F: πίνακας δυνάμεων
  double *h_C,*h_V,*h_F;

  // Μεταβλητές device (αντίστοιχη λειτουργία όπως στον host)
  int *d_N,*d_loop;
  double *d_C,*d_F,*d_V;

  // start/stop: μεταβλητές μέτρησης χρόνου
  float time;
  hipEvent_t start, stop;

  // Εισαγωγή δεδομένων απο τον χρήστη
  do
  {
    printf("\nInsert iterations: ");
    scanf("%d",&h_loop);
  }while(h_loop<10);

  do
  {
    printf("\nInsert body count: ");
    scanf("%d",&h_N);
  }while(h_N<2 || h_N>10);

  // Δέσμευση μνήμης για τον host
  h_C=(double*)malloc(h_N*3*sizeof(double));
  h_F=(double*)malloc(h_N*3*sizeof(double));
  h_V=(double*)malloc(h_N*3*sizeof(double));

  if(h_C==NULL || h_F==NULL || h_V==NULL)
  {
    printf("\nAllocation in CPU failed");
    return -1;
  }

  // Αρχικοπόιηση μευαβλητών
  for(i=0;i<h_N*3;i+=3)
  {
    // Ορισμός αρχικής θέσης
    do
    {
      u=(double)rand()/(double)RAND_MAX;
      v=(double)rand()/(double)RAND_MAX;

      x1=sqrt(8/M_E)*(v-0.5)/u;
    }while(pow(x1,2)>=-4*log(u) || x1<0 || x1>1);

    r=pow(pow(x1,-0.667)-1,-0.5);

    theta=(double)rand()/(double)RAND_MAX*M_PI;
    phi=(double)rand()/(double)RAND_MAX*2*M_PI;

    h_C[i]=r*sin(theta)*cos(phi);
    h_C[i+1]=r*sin(theta)*sin(phi);
    h_C[i+2]=r*cos(theta);

    // Ορισμός αρχικής ταχύτητας
    do
    {
      x4=rand();
      x5=rand();
    }while(0.1*x5>=pow(x4,2)*pow(1-pow(x4,2),3.5));

    vesc=sqrt(2)*pow(1+pow(r,2),-0.25);
    vel=vesc/x4;

    theta=(double)rand()/(double)RAND_MAX*M_PI;
    phi=(double)rand()/(double)RAND_MAX*2*M_PI;

    h_V[i]=vel*sin(theta)*cos(phi);
    h_V[i+1]=vel*sin(theta)*sin(phi);
    h_V[i+2]=vel*cos(theta);
  }

  for(i=0;i<h_N*3;i++)
  {
    h_F[i]=0.0;
  }

  // Αρχηκή θέση σωμάτων
  for(i=0;i<h_N*3;i+=3)
  {
    printf("\nMass %d\tC[x]: %.3f\tC[y]: %.3f\tC[z]: %.3f",i/3,h_C[i],h_C[i+1],h_C[i+2]);
  }

  printf("\n\n");

  // Έναρξη χρονομέτρησης
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Δέσμευση μνήμης για το device
  hipMalloc((void**)&d_loop,sizeof(int));
  hipMalloc((void**)&d_N,sizeof(int));

  hipMalloc((void**)&d_C,h_N*3*sizeof(double));
  hipMalloc((void**)&d_F,h_N*3*sizeof(double));
  hipMalloc((void**)&d_V,h_N*3*sizeof(double));

  if(d_loop==NULL || d_N==NULL || d_C==NULL || d_F==NULL || d_V==NULL)
  {
    printf("\nAllocation in GPU failed");
    return -1;
  }

  // Αντιγραφή δεδομένων απο το host στο device
  hipMemcpy(d_loop,&h_loop,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_N,&h_N,sizeof(int),hipMemcpyHostToDevice);

  hipMemcpy(d_C,h_C,h_N*3*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(d_F,h_F,h_N*3*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(d_V,h_V,h_N*3*sizeof(double),hipMemcpyHostToDevice);

  // Μεταβλητές παραλληλίας κάρτας γραφικών
  // threadsPerBlock: τα threads που θα εκτελεστουν σε κάθε block
  // blocksPerGrid: το πλήθος των blocks που περιέχονται μέσα σε ένα grid
  int threadsPerBlock = 256;
  int blocksPerGrid=(h_N+threadsPerBlock-1)/threadsPerBlock;

  newPos<<<blocksPerGrid,threadsPerBlock>>>(d_loop,d_N,d_C,d_V,d_F);

  // Αντιγραφή αποτελεσμάτων
  hipMemcpy(h_C,d_C,h_N*3*sizeof(double),hipMemcpyDeviceToHost);

  // Τέλος χρονομέτρησης
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  // Τελική θέση σωμάτων
  for(i=0;i<h_N*3;i+=3)
  {
    printf("\nMass %d\tC[x]: %.3f\tC[y]: %.3f\tC[z]: %.3f",i/3,h_C[i],h_C[i+1],h_C[i+2]);
  }
  printf("\n\nTime required is: %.3f\n",time*pow(10,-3));

  // Αποδέσμευση μνήμης host
  free(h_C);
  free(h_F);
  free(h_V);

  // Αποδέσμευση μνήμης device
  hipFree(d_loop);
  hipFree(d_N);
  hipFree(d_C);
  hipFree(d_F);
  hipFree(d_V);

  return 0;
}
